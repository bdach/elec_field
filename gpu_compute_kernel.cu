#include "hip/hip_runtime.h"
#include "structs.h"
#include "math.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define MIN_INTENSITY 1e-10
#define MAX_INTENSITY 1e10
#define THREAD_COUNT 1024

extern "C" void run_kernel(const point_charge_t *charges,
		const int charge_count,
		const bounds_t *bounds,
		uint32_t *result);

__global__ void calculate_intensity(const point_charge_t* charges,
		const bounds_t* bounds,
		double* result) {
	const double k = 8.99e-9; // Coulomb's constant
	point_charge_t charge = charges[threadIdx.x];
	double x_scaled = bounds->x_min + blockIdx.x * bounds->x_scale / (double)gridDim.x;
	double y_scaled = bounds->y_min + blockIdx.y * bounds->y_scale / (double)gridDim.y;
	double dx = charge.x - x_scaled;
	double dy = charge.y - y_scaled;
	double r = sqrt(dx * dx + dy * dy);
	double intensity = k * charge.charge / r;
	unsigned long offset = blockDim.x * (gridDim.x * blockIdx.y + blockIdx.x);
	result[2 * offset + threadIdx.x] = intensity * dx / r;
	result[2 * offset + blockDim.x + threadIdx.x] = intensity * dy / r;
}

__global__ void add_intensities(double *g_idata, 
		double *g_odata)
{
	extern __shared__ double sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[tid];
}

__global__ void total_intensity(double *g_idata,
		double *g_odata,
		unsigned int n)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		double x = g_idata[2 * i];
		double y = g_idata[2 * i + 1];
		g_odata[i] = fmax(fmin(sqrt(x * x + y * y), MAX_INTENSITY), MIN_INTENSITY);
	}
}

__global__ void get_min_intensity(double *g_idata,
		double *g_odata)
{
	extern __shared__ double sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s)
			sdata[tid] = fmin(sdata[tid], sdata[tid + s]);
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[tid];
}

__global__ void get_max_intensity(double *g_idata,
		double *g_odata)
{
	extern __shared__ double sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i];

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s)
			sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[tid];
}

__global__ void intensity_to_color(double *g_idata,
		uint32_t *g_odata,
		const double min,
		const double max)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	double diff = max - min;
	double log = log10(g_idata[i]);
	double scaled = (log - min) / diff;
	double hue = (1 - scaled) * 300;
	double h_prim = hue / 60.0;
	double f_x = 1 - fabs(fmod(h_prim, 2.0) - 1);
	uint8_t x = (uint8_t)(f_x * 0xFF);
	unsigned int rounded_h = (unsigned int) h_prim + 1;
	g_odata[i] = 0;
	g_odata[i] |= x << ((rounded_h % 3) * 8);
	g_odata[i] |= 0xff << (8 * (2 - ((rounded_h / 2) % 3)));
}

extern "C" void run_kernel(const point_charge_t *charges,
		const int charge_count,
		const bounds_t *bounds,
		uint32_t *result)
{
	const unsigned int charges_size = charge_count * sizeof(point_charge_t);
	const unsigned int bounds_size = sizeof(bounds_t);
	const unsigned long result_size = 2 * sizeof(double) * charge_count * bounds->width * bounds->height;
	const unsigned long reduced_size = sizeof(uint32_t) * bounds->width * bounds->height;


	point_charge_t *d_charges;
	checkCudaErrors(hipMalloc((void **)&d_charges, charges_size));
	checkCudaErrors(hipMemcpy(d_charges, charges, charges_size, hipMemcpyHostToDevice));

	bounds_t *d_bounds;
	checkCudaErrors(hipMalloc((void**)&d_bounds, bounds_size));
	checkCudaErrors(hipMemcpy(d_bounds, bounds, bounds_size, hipMemcpyHostToDevice));

	double *d_result_vec;
	checkCudaErrors(hipMalloc((void**)&d_result_vec, result_size));

	dim3 charge_intensity_grid(bounds->width, bounds->height, 1);
	dim3 threads(charge_count, 1, 1);

	calculate_intensity<<< charge_intensity_grid, threads >>>(d_charges, d_bounds, d_result_vec);
	getLastCudaError("Intensity calculation failed");

	dim3 component_intensity_grid(2 * bounds->width * bounds->height, 1, 1);
	unsigned int smem = sizeof(double) * charge_count;
	add_intensities<<< component_intensity_grid, threads, smem >>>(d_result_vec, d_result_vec);
	getLastCudaError("Intensity reduction failed");

	int block_count = bounds->width * bounds->height / THREAD_COUNT;
	dim3 max_thread_grid(block_count, 1, 1);
	total_intensity<<< max_thread_grid, THREAD_COUNT >>>(d_result_vec, d_result_vec, bounds->width * bounds->height);
	getLastCudaError("Total intensity calculation failed");

	double min, max;
	double *d_minmax_temp_buf;
	checkCudaErrors(hipMalloc((void**)&d_minmax_temp_buf, sizeof(double) * block_count));

	smem = sizeof(double) * THREAD_COUNT;
	get_min_intensity<<< max_thread_grid, THREAD_COUNT, smem >>>(d_result_vec, d_minmax_temp_buf);
	getLastCudaError("Minimum: first iteration failed");
	get_min_intensity<<< 1, THREAD_COUNT, smem >>>(d_minmax_temp_buf, d_minmax_temp_buf);
	getLastCudaError("Minimum: second iteration failed");
	checkCudaErrors(hipMemcpy(&min, d_minmax_temp_buf, sizeof(double), hipMemcpyDeviceToHost));

	get_max_intensity<<< max_thread_grid, THREAD_COUNT, smem >>>(d_result_vec, d_minmax_temp_buf);
	getLastCudaError("Maximum: first iteration failed");
	get_max_intensity<<< 1, THREAD_COUNT, smem >>>(d_minmax_temp_buf, d_minmax_temp_buf);
	getLastCudaError("Maximum: second iteration failed");
	checkCudaErrors(hipMemcpy(&max, d_minmax_temp_buf, sizeof(double), hipMemcpyDeviceToHost));

	min = log10(fmax(min, MIN_INTENSITY));
	max = log10(fmin(max, MAX_INTENSITY));

	intensity_to_color<<< max_thread_grid, THREAD_COUNT >>>(d_result_vec, (uint32_t*)d_result_vec, min, max);
	getLastCudaError("Conversion to color failed");

	checkCudaErrors(hipMemcpy(result, (uint32_t*)d_result_vec, reduced_size, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_charges));
	checkCudaErrors(hipFree(d_bounds));
	checkCudaErrors(hipFree(d_result_vec));
	checkCudaErrors(hipFree(d_minmax_temp_buf));
}

